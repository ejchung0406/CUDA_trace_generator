#include "hip/hip_runtime.h"
/* Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <assert.h>
#include <stdint.h>
#include <stdio.h>
#include <unistd.h>
#include <map>
#include <sstream>
#include <string>
#include <unordered_set>
#include <unordered_map>
#include <pthread.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <thread>
#include <mutex>
#include <queue>
#include <set>
#include <condition_variable>
#include <functional>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <zlib.h>
#include <sys/stat.h>
#include <fstream>

#define MAX_GPU_SRC_NUM 4
#define MAX_GPU_DST_NUM 4

/* every tool needs to include this once */
#include "nvbit_tool.h"

/* nvbit interface file */
#include "nvbit.h"

/* for channel */
#include "utils/channel.hpp"

/* contains definition of the mem_access_t structure */
#include "common.h"
#include "mem_trace.h"

#define HEX(x)                                                            \
    "0x" << std::setfill('0') << std::setw(16) << std::hex << (uint64_t)x \
         << std::dec

#define CHANNEL_SIZE (1l << 20)

struct CTXstate {
    /* context id */
    int id;

    /* Channel used to communicate from GPU to CPU receiving thread */
    ChannelDev* channel_dev;
    ChannelHost channel_host;
};

/* lock */
pthread_mutex_t mutex;

/* map to store context state */
std::unordered_map<hipCtx_t, CTXstate*> ctx_state_map;

/* skip flag used to avoid re-entry on the nvbit_callback when issuing
 * flush_channel kernel call */
bool skip_callback_flag = false;

/* global control variables for this tool */
uint32_t instr_begin_interval = 0;
uint32_t instr_end_interval = UINT32_MAX;
int verbose = 0;

/* opcode to id map and reverse map  */
std::map<std::string, int> opcode_to_id_map;
std::map<int, std::string> id_to_opcode_map;

/* OPCODE list */
std::vector<std::string> FP_LIST;
std::vector<std::string> LD_LIST;
std::vector<std::string> ST_LIST;
// const std::string GPU_NVBIT_OPCODE[];
// const std::string CF_TYPE[];

/* grid launch id, incremented at every launch */
uint64_t grid_launch_id = 0;

/* # of workers for file i/o? */
const size_t num_threads = 8;

/* # of warps */
const size_t num_warps = 4096 / 32;

/* Trace file path */
std::string trace_path = "/home/echung67/nvbit_release/tools/main/trace/";
std::string insts_path = "/home/echung67/nvbit_release/tools/main/insts/";

/* Warp ids */
std::priority_queue<int, std::vector<int>, std::greater<int>> warp_ids;
std::set<int> warp_ids_s;

class ThreadPool {
public:
    ThreadPool(size_t num_threads) : stop(false) {
        for (size_t i = 0; i < num_threads; i++) {
            threads.emplace_back([this] {
                while (true) {
                    std::function<void()> task;
                    {
                        std::unique_lock<std::mutex> lock(mutex);
                        cv.wait(lock, [this] { return stop || !tasks.empty(); });
                        if (stop && tasks.empty()) {
                            return;
                        }
                        task = std::move(tasks.front());
                        tasks.pop();
                    }
                    task();
                }
            });
        }
    }

    ~ThreadPool() {
        {
            std::unique_lock<std::mutex> lock(mutex);
            stop = true;
        }
        cv.notify_all();
        for (std::thread& thread : threads) {
            thread.join();
        }
    }

    template<class F, class... Args>
    void enqueue(F&& f, Args&&... args) {
        {
            std::unique_lock<std::mutex> lock(mutex);
            tasks.emplace([=] { f(args...); });
        }
        cv.notify_one();
    }

private:
    std::vector<std::thread> threads;
    std::queue<std::function<void()>> tasks;
    std::mutex mutex;
    std::condition_variable cv;
    bool stop;
};

bool is_fp(std::string opcode){
    std::size_t dot_pos = opcode.find('.');
    std::string opcode_short = opcode.substr(0, dot_pos);

    auto it = std::find(FP_LIST.begin(), FP_LIST.end(), opcode_short);
    return (it != FP_LIST.end())? true : false;
}

bool is_ld(std::string opcode){
    std::size_t dot_pos = opcode.find('.');
    std::string opcode_short = opcode.substr(0, dot_pos);

    auto it = std::find(LD_LIST.begin(), LD_LIST.end(), opcode_short);
    return (it != LD_LIST.end()) ? true : false;
}

bool is_st(std::string opcode){
    std::size_t dot_pos = opcode.find('.');
    std::string opcode_short = opcode.substr(0, dot_pos);

    auto it = std::find(ST_LIST.begin(), ST_LIST.end(), opcode_short);
    return (it != ST_LIST.end()) ? true : false;
}

bool file_exists(const std::string& file_path) {
    std::ifstream f(file_path);
    return f.good();
}

// not sure..
std::string cf_type(std::string opcode){ 
    // NOT_CF,  //!< not a control flow instruction
    // CF_BR,  //!< an unconditional branch
    // CF_CBR,  //!< a conditional branch
    // CF_CALL,  //!< a call
    // // below this point are indirect cfs
    // CF_IBR,  //!< an indirect branch // non conditional
    // CF_ICALL,  //!< an indirect call
    // CF_ICO,  //!< an indirect jump to co-routine
    // CF_RET,  //!< a return
    // CF_MITE,  //!< alpha PAL, micro-instruction assited instructions

    std::size_t dot_pos = opcode.find('.');
    std::string opcode_short = opcode.substr(0, dot_pos);
    if (opcode_short == "JMP")
        return "CF_BR";
    else if (opcode_short == "BRA")
        return "CF_CBR";
    else if (opcode_short == "RET")
        return "CF_RET";
    else 
        return "NOT_CF";
}

uint8_t num_dst_reg(mem_access_t* ma){
    std::string opcode = id_to_opcode_map[ma->opcode_id];
    std::size_t dot_pos = opcode.find('.');
    std::string opcode_short = opcode.substr(0, dot_pos);
    if (opcode_short == "BRA" || opcode_short == "EXIT" || is_st(opcode))
        return 0;
    else 
        return 1;
}

void src_reg(mem_access_t* ma, uint16_t* src_reg_){
    for(int i=num_dst_reg(ma), j=0; i<ma->num_regs; i++, j++){
        src_reg_[j] = ma->reg_id[i];
    }
    return;
}

void dst_reg(mem_access_t* ma, uint16_t* dst_reg_){
    for(int i=0; i<num_dst_reg(ma); i++){
        dst_reg_[i] = ma->reg_id[i];
    }
    return;
}

void nvbit_at_init() {
    setenv("CUDA_MANAGED_FORCE_DEVICE_ALLOC", "1", 1);
    GET_VAR_INT(
        instr_begin_interval, "INSTR_BEGIN", 0,
        "Beginning of the instruction interval where to apply instrumentation");
    GET_VAR_INT(
        instr_end_interval, "INSTR_END", UINT32_MAX,
        "End of the instruction interval where to apply instrumentation");
    GET_VAR_INT(verbose, "TOOL_VERBOSE", 0, "Enable verbosity inside the tool");
    std::string pad(100, '-');
    printf("%s\n", pad.c_str());

    /* set mutex as recursive */
    pthread_mutexattr_t attr;
    pthread_mutexattr_init(&attr);
    pthread_mutexattr_settype(&attr, PTHREAD_MUTEX_RECURSIVE);
    pthread_mutex_init(&mutex, &attr);

    // std::ifstream file("insts.txt");
    // std::vector<std::string> GPU_OPCODE_LIST;
    // std::string line;
    std::ifstream file_fl(insts_path + "floating.txt");
    std::string line_fl;
    std::ifstream file_ld(insts_path + "load.txt");
    std::string line_ld;
    std::ifstream file_st(insts_path + "store.txt");
    std::string line_st;
    // while (std::getline(file, line)) {
    //     GPU_OPCODE_LIST.push_back(line);
    // }
    while (std::getline(file_fl, line_fl)) {
        FP_LIST.push_back(line_fl);
    }
    while (std::getline(file_ld, line_ld)) {
        LD_LIST.push_back(line_ld);
    }
    while (std::getline(file_st, line_st)) {
        ST_LIST.push_back(line_st);
    }
    // for (const auto& l : LD_LIST) {
    //     std::cout << l << std::endl;
    // }

    ThreadPool pool(num_threads);

    std::string command = "rm " + trace_path + "*.*";
    int status = system(command.c_str());
    if (status == 0) {
        std::cout << "rm command executed successfully" << std::endl;
    } else {
        std::cout << "rm command failed to execute" << std::endl;
    }

    std::ofstream file_info_trace(trace_path + "trace.txt");
    file_info_trace << "GPU" << std::endl;
    file_info_trace << "nvbit" << std::endl;
    file_info_trace << "14" << std::endl; // GPU Trace version (??)
    file_info_trace << "6" << std::endl; // Max blocks per core (?? hardcoded..)
    file_info_trace << num_warps << std::endl; // Total number of warps (hardcoded..)
    file_info_trace.close();
}

/* Set used to avoid re-instrumenting the same functions multiple times */
std::unordered_set<hipFunction_t> already_instrumented;

void instrument_function_if_needed(hipCtx_t ctx, hipFunction_t func) {
    assert(ctx_state_map.find(ctx) != ctx_state_map.end());
    CTXstate* ctx_state = ctx_state_map[ctx];

    /* Get related functions of the kernel (device function that can be
     * called by the kernel) */
    std::vector<hipFunction_t> related_functions =
        nvbit_get_related_functions(ctx, func);

    /* add kernel itself to the related function vector */
    related_functions.push_back(func);

    /* iterate on function */
    for (auto f : related_functions) {
        /* "recording" function was instrumented, if set insertion failed
         * we have already encountered this function */
        if (!already_instrumented.insert(f).second) {
            continue;
        }

        /* get vector of instructions of function "f" */
        const std::vector<Instr*>& instrs = nvbit_get_instrs(ctx, f);

        if (verbose) {
            printf(
                "MEMTRACE: CTX %p, Inspecting hipFunction_t %p name %s at address "
                "0x%lx\n",
                ctx, f, nvbit_get_func_name(ctx, f), nvbit_get_func_addr(f));
        }

        uint32_t cnt = 0;
        /* iterate on all the static instructions in the function */
        for (auto instr : instrs) {
            // if (cnt < instr_begin_interval || cnt >= instr_end_interval ||
            //     instr->getMemorySpace() == InstrType::MemorySpace::NONE ||
            //     instr->getMemorySpace() == InstrType::MemorySpace::CONSTANT) {
            //     cnt++;
            //     continue;
            // }
            if (cnt < instr_begin_interval || cnt >= instr_end_interval) {
                cnt++;
                continue;
            }
            if (verbose) {
                instr->printDecoded();
            }

            if (opcode_to_id_map.find(instr->getOpcode()) ==
                opcode_to_id_map.end()) {
                int opcode_id = opcode_to_id_map.size();
                opcode_to_id_map[instr->getOpcode()] = opcode_id;
                id_to_opcode_map[opcode_id] = std::string(instr->getOpcode());
            }

            int opcode_id = opcode_to_id_map[instr->getOpcode()];
            std::vector<int> reg_num_list;
            // int mref_idx = 0;
            /* iterate on the operands */
            for (int i = 0; i < instr->getNumOperands(); i++) {
                /* get the operand "i" */
                const InstrType::operand_t* op = instr->getOperand(i);

                /* count # of regs */
                if (op->type == InstrType::OperandType::REG || 
                    op->type == InstrType::OperandType::PRED) {
                    for (int reg_idx = 0; reg_idx < instr->getSize() / 4; reg_idx++) {
                        reg_num_list.push_back(op->u.reg.num + reg_idx);
                    }
                }
            }

                // if (op->type == InstrType::OperandType::MREF) {
                //     /* insert call to the instrumentation function with its
                //      * arguments */
                //     nvbit_insert_call(instr, "instrument_mem", IPOINT_BEFORE);
                //     /* predicate value */
                //     nvbit_add_call_arg_guard_pred_val(instr);
                //     /* opcode id */
                //     nvbit_add_call_arg_const_val32(instr, opcode_id);
                //     /* memory reference 64 bit address */
                //     nvbit_add_call_arg_mref_addr64(instr, mref_idx);
                //     mref_idx++;
                // } else {
            nvbit_insert_call(instr, "instrument_else", IPOINT_BEFORE);
            nvbit_add_call_arg_guard_pred_val(instr);
            nvbit_add_call_arg_const_val32(instr, opcode_id);
            // }
            /* add "space" for kernel function pointer that will be set
                    * at launch time (64 bit value at offset 0 of the dynamic
                    * arguments)*/
            nvbit_add_call_arg_launch_val64(instr, 0);
            /* add pointer to channel_dev*/
            nvbit_add_call_arg_const_val64(instr, (uint64_t)ctx_state->channel_dev);
            /* instruction size */
            nvbit_add_call_arg_const_val32(instr, 4); // 32bit instructions?
            /* PC address */
            nvbit_add_call_arg_const_val64(instr, nvbit_get_func_addr(func) + 8 * instr->getOffset());
            // std::cout << instr->getSass() << " func addr: " << nvbit_get_func_addr(func) << " offset: " << instr->getOffset() << std::endl;
            /* MEM access address / reconv(??) address */
            nvbit_add_call_arg_mref_addr64(instr);
            /* MEM access size */
            nvbit_add_call_arg_const_val32(instr, (uint8_t)instr->getSize()); // i'm not sure what this getSize() function exactly does.. is it for getting instruction size or mem_access size?
            /* MEM addr space */
            nvbit_add_call_arg_const_val32(instr, (uint8_t)instr->getMemorySpace());
            /* how many register values are passed next */
            nvbit_add_call_arg_const_val32(instr, reg_num_list.size());
            std::cout << instr->getSass() << ", reg_num: " << reg_num_list.size() << std::endl;
            for (int num : reg_num_list) {
                /* last parameter tells it is a variadic parameter passed to
                * the instrument function record_reg_val() */
                nvbit_add_call_arg_const_val32(instr, num, true);
                std::cout << "num: " << num << " ";
            }
            std::cout << std::endl;
            cnt++;
        }
    }
}

__global__ void flush_channel(ChannelDev* ch_dev) {
    /* set a CTA id = -1 to indicate communication thread that this is the
     * termination flag */
    mem_access_t ma;
    ma.cta_id_x = -1;
    ch_dev->push(&ma, sizeof(mem_access_t));
    /* flush channel */
    ch_dev->flush();
}

void nvbit_at_cuda_event(hipCtx_t ctx, int is_exit, nvbit_api_cuda_t cbid,
                         const char* name, void* params, hipError_t* pStatus) {
    pthread_mutex_lock(&mutex);

    /* we prevent re-entry on this callback when issuing CUDA functions inside
     * this function */
    if (skip_callback_flag) {
        pthread_mutex_unlock(&mutex);
        return;
    }
    skip_callback_flag = true;

    assert(ctx_state_map.find(ctx) != ctx_state_map.end());
    CTXstate* ctx_state = ctx_state_map[ctx];

    if (cbid == API_CUDA_cuLaunchKernel_ptsz ||
        cbid == API_CUDA_cuLaunchKernel) {
        cuLaunchKernel_params* p = (cuLaunchKernel_params*)params;

        /* Make sure GPU is idle */
        hipDeviceSynchronize();
        assert(hipGetLastError() == hipSuccess);

        if (!is_exit) {
            /* instrument */
            instrument_function_if_needed(ctx, p->f);

            int nregs = 0;
            CUDA_SAFECALL(
                hipFuncGetAttribute(&nregs, HIP_FUNC_ATTRIBUTE_NUM_REGS, p->f));

            int shmem_static_nbytes = 0;
            CUDA_SAFECALL(
                hipFuncGetAttribute(&shmem_static_nbytes,
                                   HIP_FUNC_ATTRIBUTE_SHARED_SIZE_BYTES, p->f));

            /* get function name and pc */
            const char* func_name = nvbit_get_func_name(ctx, p->f);
            uint64_t pc = nvbit_get_func_addr(p->f);

            /* set grid launch id at launch time */
            nvbit_set_at_launch(ctx, p->f, &grid_launch_id, sizeof(uint64_t));
            /* increment grid launch id for next launch */
            grid_launch_id++;

            /* enable instrumented code to run */
            nvbit_enable_instrumented(ctx, p->f, true);

            printf(
                "MEMTRACE: CTX 0x%016lx - LAUNCH - Kernel pc 0x%016lx - Kernel "
                "name %s - grid launch id %ld - grid size %d,%d,%d - block "
                "size %d,%d,%d - nregs %d - shmem %d - cuda stream id %ld\n",
                (uint64_t)ctx, pc, func_name, grid_launch_id, p->gridDimX,
                p->gridDimY, p->gridDimZ, p->blockDimX, p->blockDimY,
                p->blockDimZ, nregs, shmem_static_nbytes + p->sharedMemBytes,
                (uint64_t)p->hStream);
        } else {
            // save to file
            std::cout << std::endl;
        }
    }
    skip_callback_flag = false;
    pthread_mutex_unlock(&mutex);
}

void* recv_thread_fun(void* args) {
    hipCtx_t ctx = (hipCtx_t)args;
    ThreadPool pool(num_threads);

    pthread_mutex_lock(&mutex);
    /* get context state from map */
    assert(ctx_state_map.find(ctx) != ctx_state_map.end());
    CTXstate* ctx_state = ctx_state_map[ctx];

    ChannelHost* ch_host = &ctx_state->channel_host;
    pthread_mutex_unlock(&mutex);
    char* recv_buffer = (char*)malloc(CHANNEL_SIZE);

    bool done = false;
    while (!done) {
        /* receive buffer from channel */
        uint32_t num_recv_bytes = ch_host->recv(recv_buffer, CHANNEL_SIZE);
        if (num_recv_bytes > 0) {
            uint32_t num_processed_bytes = 0;
            while (num_processed_bytes < num_recv_bytes) {
                mem_access_t* ma =
                    (mem_access_t*)&recv_buffer[num_processed_bytes];

                /* when we receive a CTA_id_x it means all the kernels
                 * completed, this is the special token we receive from the
                 * flush channel kernel that is issues at the end of the
                 * context */
                if (ma->cta_id_x == -1) {
                    done = true;
                    break;
                }

                std::stringstream ss;
                // ss << "CTX " << HEX(ctx) << " - grid_launch_id "
                //    << ma->grid_launch_id << " - CTA " << ma->cta_id_x << ","
                //    << ma->cta_id_y << "," << ma->cta_id_z << " - warp "
                //    << ma->warp_id << " - " << id_to_opcode_map[ma->opcode_id]
                //    << " - ";

                // for (int i = 0; i < 32; i++) {
                //     ss << HEX(ma->addrs[i]) << " ";
                // }

                std::string filename = "trace_" + std::to_string(ma->warp_id) + ".txt";
                std::string filename_raw = "trace_" + std::to_string(ma->warp_id) + ".raw";
                std::string opcode = id_to_opcode_map[ma->opcode_id];

                std::size_t dot_pos = opcode.find('.');
                std::string opcode_short = opcode.substr(0, dot_pos);
                uint8_t opcode_int = 255;
                auto it = std::find(std::begin(GPU_NVBIT_OPCODE), std::end(GPU_NVBIT_OPCODE), opcode_short);
                if (it != std::end(GPU_NVBIT_OPCODE)) {
                    opcode_int = (uint8_t)std::distance(std::begin(GPU_NVBIT_OPCODE), it);
                }
                uint8_t cf_type_int = 255;
                it = std::find(std::begin(CF_TYPE), std::end(CF_TYPE), cf_type(opcode));
                if (it != std::end(CF_TYPE)) {
                    cf_type_int = (uint8_t)std::distance(std::begin(CF_TYPE), it);
                }
                uint8_t num_dst_reg_ = num_dst_reg(ma);
                uint8_t num_src_reg_ = ma->num_regs - num_dst_reg_;
                uint16_t src_reg_[MAX_GPU_SRC_NUM];
                uint16_t dst_reg_[MAX_GPU_DST_NUM];
                memset(src_reg_, 0, sizeof(src_reg_));
                memset(dst_reg_, 0, sizeof(dst_reg_));
                src_reg(ma, src_reg_);
                dst_reg(ma, dst_reg_);
                uint8_t size = ma->size; // always 4? 8?
                uint32_t active_mask = ma->active_mask;
                uint32_t br_taken_mask = 0; // should be added soon
                uint64_t func_addr = ma->func_addr;
                uint64_t br_target_addr = 0; // should be added soon
                uint64_t mem_addr = ma->mem_addr; // or m_reconv_inst_addr
                uint8_t mem_access_size = ma->mem_access_size; // or m_barrier_id
                uint16_t m_num_barrier_threads = 0; // should added soon
                uint8_t m_addr_space = ma->m_addr_space; // or m_level (memory barrier level)
                std::string m_addr_space_ = MemorySpaceStr[m_addr_space];
                uint8_t m_cache_level = 0; // should be added soon
                uint8_t m_cache_operator = 0; // should be added soon

                // I don't know why but gzopen and nvbit don't work together.......
                // I guess it will need another program that compresses every .raw file with zlib
                // std::string filepath = trace_path + filename_raw;
                // gzFile file_raw = nullptr;
                // if (file_exists(filepath)){
                //     file_raw = gzopen(filepath.c_str(), "wb");
                // }
                // gzFile file_raw = (fileExists(filepath)) ? gzopen(filepath.c_str(), "ab") : gzopen(filepath.c_str(), "wb");

                if(warp_ids_s.find(ma->warp_id) == warp_ids_s.end()) {
                    warp_ids.push(ma->warp_id);
                    warp_ids_s.insert(ma->warp_id);
                }

                pool.enqueue([filename, filename_raw, opcode, opcode_int, cf_type_int, num_src_reg_, num_dst_reg_, size, 
                    src_reg_, dst_reg_, active_mask, br_taken_mask, func_addr, br_target_addr, mem_addr, 
                    mem_access_size, m_num_barrier_threads, m_addr_space_, m_cache_level, m_cache_operator] {
                    std::ofstream file(trace_path + filename, std::ios_base::app);
                    std::ofstream file_raw(trace_path + filename_raw, std::ios::binary | std::ios_base::app);

                    file << opcode << std::endl;
                    file << is_fp(opcode) << std::endl;
                    file << is_ld(opcode) << std::endl;
                    file << cf_type(opcode) << std::endl;
                    file << (int)num_src_reg_ << std::endl;
                    file << (int)num_dst_reg_ << std::endl;
                    file << src_reg_[0] << std::endl;
                    file << src_reg_[1] << std::endl;
                    file << src_reg_[2] << std::endl;
                    file << src_reg_[3] << std::endl;
                    file << dst_reg_[0] << std::endl;
                    file << dst_reg_[1] << std::endl;
                    file << dst_reg_[2] << std::endl;
                    file << dst_reg_[3] << std::endl;
                    file << (int)size << std::endl;
                    file << std::hex << active_mask << std::endl;
                    file << std::hex << br_taken_mask << std::endl;
                    file << std::hex << func_addr << std::endl;
                    file << std::hex << br_target_addr << std::endl; 
                    file << std::hex << mem_addr << std::endl;
                    file << (int)mem_access_size << std::endl;
                    file << (int)m_num_barrier_threads << std::endl;
                    file << m_addr_space_ << std::endl;
                    file << (int)m_cache_level << std::endl;
                    file << (int)m_cache_operator << std::endl;
                    file << std::endl;
                    file.close();
                    
                    bool is_fp_ = is_fp(opcode);
                    bool is_ld_ = is_ld(opcode);
                    file_raw.write(reinterpret_cast<const char*>(&opcode_int), sizeof(opcode_int));
                    file_raw.write(reinterpret_cast<const char*>(&is_fp_), sizeof(bool));
                    file_raw.write(reinterpret_cast<const char*>(&is_ld_), sizeof(bool));
                    file_raw.write(reinterpret_cast<const char*>(&cf_type_int), sizeof(cf_type_int));
                    file_raw.write(reinterpret_cast<const char*>(&num_src_reg_), sizeof(num_src_reg_));
                    file_raw.write(reinterpret_cast<const char*>(&num_dst_reg_), sizeof(num_dst_reg_));
                    file_raw.write(reinterpret_cast<const char*>(src_reg_), num_src_reg_ * sizeof(uint16_t));
                    file_raw.write(reinterpret_cast<const char*>(dst_reg_), num_dst_reg_ * sizeof(uint16_t));
                    file_raw.write(reinterpret_cast<const char*>(&size), sizeof(size));
                    file_raw.write(reinterpret_cast<const char*>(&active_mask), sizeof(active_mask));
                    file_raw.write(reinterpret_cast<const char*>(&br_taken_mask), sizeof(br_taken_mask));
                    file_raw.write(reinterpret_cast<const char*>(&func_addr), sizeof(func_addr));
                    file_raw.write(reinterpret_cast<const char*>(&br_target_addr), sizeof(br_target_addr));
                    file_raw.write(reinterpret_cast<const char*>(&mem_addr), sizeof(mem_addr));
                    file_raw.write(reinterpret_cast<const char*>(&mem_access_size), sizeof(mem_access_size));
                    file_raw.write(reinterpret_cast<const char*>(&m_num_barrier_threads), sizeof(m_num_barrier_threads));
                    file_raw.write(reinterpret_cast<const char*>(&m_addr_space_), sizeof(m_addr_space_));
                    file_raw.write(reinterpret_cast<const char*>(&m_cache_level), sizeof(m_cache_level));
                    file_raw.write(reinterpret_cast<const char*>(&m_cache_operator), sizeof(m_cache_operator));
                    file_raw.close();

                    // gzopen() doesn't work.. so i will just leave these commented
                    // gzwrite(file_raw, &opcode_int, sizeof(opcode_int));
                    // gzwrite(file_raw, &is_fp_, sizeof(bool));
                    // gzwrite(file_raw, &is_ld_, sizeof(bool));
                    // gzwrite(file_raw, &cf_type_int, sizeof(cf_type_int));
                    // gzwrite(file_raw, &num_src_reg_, sizeof(num_src_reg_));
                    // gzwrite(file_raw, &num_dst_reg_, sizeof(num_dst_reg_));
                    // gzwrite(file_raw, src_reg_, num_src_reg_ * sizeof(uint16_t));
                    // gzwrite(file_raw, dst_reg_, num_dst_reg_ * sizeof(uint16_t));
                    // gzwrite(file_raw, &size, sizeof(size));
                    // gzwrite(file_raw, &active_mask, sizeof(active_mask));
                    // gzwrite(file_raw, &br_taken_mask, sizeof(br_taken_mask));
                    // gzwrite(file_raw, &func_addr, sizeof(func_addr));
                    // gzwrite(file_raw, &br_target_addr, sizeof(br_target_addr));
                    // gzwrite(file_raw, &mem_addr, sizeof(mem_addr));
                    // gzwrite(file_raw, &mem_access_size, sizeof(mem_access_size));
                    // gzwrite(file_raw, &m_num_barrier_threads, sizeof(m_num_barrier_threads));
                    // gzwrite(file_raw, &m_addr_space_, sizeof(m_addr_space_));
                    // gzwrite(file_raw, &m_cache_level, sizeof(m_cache_level));
                    // gzwrite(file_raw, &m_cache_operator, sizeof(m_cache_operator));
                    // gzclose(file_raw);
                });
                num_processed_bytes += sizeof(mem_access_t);
            }
        }
    }
    // Print the elements in the heap in order
    std::ofstream file_info_trace(trace_path + "trace.txt", std::ios_base::app);
    while (!warp_ids.empty()) {
        file_info_trace << warp_ids.top() << " " << "0" << std::endl;
        warp_ids.pop();
    }
    file_info_trace.close();

    free(recv_buffer);
    return NULL;
}

void nvbit_at_ctx_init(hipCtx_t ctx) {
    pthread_mutex_lock(&mutex);
    if (verbose) {
        printf("MEMTRACE: STARTING CONTEXT %p\n", ctx);
    }
    CTXstate* ctx_state = new CTXstate;
    assert(ctx_state_map.find(ctx) == ctx_state_map.end());
    ctx_state_map[ctx] = ctx_state;
    hipMallocManaged(&ctx_state->channel_dev, sizeof(ChannelDev));
    ctx_state->channel_host.init((int)ctx_state_map.size() - 1, CHANNEL_SIZE,
                                 ctx_state->channel_dev, recv_thread_fun, ctx);
    nvbit_set_tool_pthread(ctx_state->channel_host.get_thread());
    pthread_mutex_unlock(&mutex);
}

void nvbit_at_ctx_term(hipCtx_t ctx) {
    pthread_mutex_lock(&mutex);

    skip_callback_flag = true;
    if (verbose) {
        printf("MEMTRACE: TERMINATING CONTEXT %p\n", ctx);
    }
    /* get context state from map */
    assert(ctx_state_map.find(ctx) != ctx_state_map.end());
    CTXstate* ctx_state = ctx_state_map[ctx];

    /* flush channel */
    flush_channel<<<1, 1>>>(ctx_state->channel_dev);
    /* Make sure flush of channel is complete */
    hipDeviceSynchronize();
    assert(hipGetLastError() == hipSuccess);

    ctx_state->channel_host.destroy(false);
    hipFree(ctx_state->channel_dev);
    skip_callback_flag = false;
    delete ctx_state;
    pthread_mutex_unlock(&mutex);
}
